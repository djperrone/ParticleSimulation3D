#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "./cuda_kernel.cuh"
#include ""
#include <iostream>

namespace Cuda {


    __global__ void vectorAdditionKernel(double* A, double* B, double* C, int arraySize) {
        // Get thread ID.
        int threadID = blockDim.x * blockIdx.x + threadIdx.x;

        // Check if thread is within array bounds.
        if (threadID < arraySize) {
            // Add a and b.
            C[threadID] = A[threadID] + B[threadID];
        }
    }

    void RunTest()
    {
        // Initialize arrays A, B, and C.
        double A[3], B[3], C[3];

        // Populate arrays A and B.
        A[0] = 5; A[1] = 8; A[2] = 3;
        B[0] = 7; B[1] = 6; B[2] = 4;

        // Sum array elements across ( C[0] = A[0] + B[0] ) into array C using CUDA.
        kernel(A, B, C, 3);

        // Print out result.
        std::cout << "C = " << C[0] << ", " << C[1] << ", " << C[2] << std::endl;
    }




    /**
     * Wrapper function for the CUDA kernel function.
     * @param A Array A.
     * @param B Array B.
     * @param C Sum of array elements A and B directly across.
     * @param arraySize Size of arrays A, B, and C.
     */
    void kernel(double* A, double* B, double* C, int arraySize) {

        // Initialize device pointers.
        double* d_A, * d_B, * d_C;

        // Allocate device memory.
        hipMalloc((void**)&d_A, arraySize * sizeof(double));
        hipMalloc((void**)&d_B, arraySize * sizeof(double));
        hipMalloc((void**)&d_C, arraySize * sizeof(double));

        // Transfer arrays a and b to device.
        hipMemcpy(d_A, A, arraySize * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, arraySize * sizeof(double), hipMemcpyHostToDevice);

        // Calculate blocksize and gridsize.
        dim3 blockSize(512, 1, 1);
        dim3 gridSize(512 / arraySize + 1, 1);

        // Launch CUDA kernel.
        vectorAdditionKernel CUDA_KERNEL(gridSize, blockSize) (d_A, d_B, d_C, arraySize);

        // Copy result array c back to host memory.
        hipMemcpy(C, d_C, arraySize * sizeof(double), hipMemcpyDeviceToHost);
    }
}